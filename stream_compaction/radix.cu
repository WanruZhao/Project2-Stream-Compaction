#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "radix.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace radix {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernRadixEArray(int n, int p, int *bdata, int *edata, const int *idata) {
            int index = blockDim.x * blockIdx.x + threadIdx.x;
            if(index >= n) return;

            int digit = idata[index] & p;
            
            edata[index] = digit ? 0 : 1;
            bdata[index] = digit ? 1 : 0;
        }

        __global__ void kernRadixDArray(int n, int p, int totalFalse, int *ddata, const int *fdata, const int *bdata) {
            int index = blockDim.x * blockIdx.x + threadIdx.x;
            if(index >= n) return;

            int f = fdata[index];
            int t = index - f + totalFalse;
            ddata[index] = bdata[index] ? t : f;
        }

        void sort(int n, int *odata, const int *idata){

            int d = ilog2ceil(n);
            int nPow = 1 << d;

            int *dev_idataPow, *dev_bdataPow, *dev_edataPow, *dev_fdataPow, *dev_ddataPow, *dev_odataPow;
            int en = 0, fn = 0;

            hipMalloc((void**)&dev_idataPow, nPow * sizeof(int));
			checkCUDAError("hipMalloc dev_idataPow failed!");
			hipMemcpy(dev_idataPow, idata, n * sizeof(int), hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy dev_idataPow failed!");
			hipMemset(dev_idataPow + n, 0, (nPow - n) * sizeof(int));
            checkCUDAError("hipMemset dev_idataPow failed!");
            
            hipMalloc((void**)&dev_edataPow, nPow * sizeof(int));
            checkCUDAError("hipMalloc dev_edataPow failed!");
            hipMemset(dev_edataPow, 0, nPow * sizeof(int));
            hipMalloc((void**)&dev_bdataPow, nPow * sizeof(int));
            checkCUDAError("hipMalloc dev_bdataPow failed!");
            hipMemset(dev_bdataPow, 0, nPow * sizeof(int));
            hipMalloc((void**)&dev_fdataPow, nPow * sizeof(int));
            checkCUDAError("hipMalloc dev_fdataPow failed!");
            hipMemset(dev_fdataPow, 0, nPow * sizeof(int));
            hipMalloc((void**)&dev_ddata, nPow * sizeof(int));
            checkCUDAError("hipMalloc dev_idata failed!");
            hipMemset(dev_ddataPow, 0, nPow * sizeof(int));
            hipMalloc((void**)&dev_odataPow, nPow * sizeof(int));
            checkCUDAError("hipMalloc dev_odata failed!");
            hipMemset(dev_odataPow, 0, nPow * sizeof(int));
            
            timer().startGpuTimer();
            
            dim3 gridDim((n + blockSize - 1) / blockSize);
            
            for(int p = 1; p <= 6; p++) {
                kernRadixEArray<<<gridDim, blockSize>>>(n, p, dev_edataPow, dev_idataPow);
                hipMemcpy(dev_fdataPow, dev_edataPow, nPow * sizeof(int), hipMemcpyDeviceToDevice);
                StreamCompaction::Efficient::scanCore(nPow, d, dev_fdataPow);
                hipMemcpy(en, dev_edataPow + n - 1, sizeof(int), hipMemcpyDeviceToHost);
                hipMemcpy(fn, dev_fdataPow + n - 1, sizeof(int), hipMemcpyDeviceToHost);
                kernRadixDArray<<<gridDim, blockSize>>>(n, p, en + fn, dev_ddataPow, dev_fdataPow, dev_bdataPow);
                StreamCompaction::Common::kernScatter<<<gridDim, blockSize>>>(n, dev_odataPow, dev_idataPow, dev_bdataPow, dev_ddataPow);
                std::swap(dev_idataPow, dev_odataPow);
            }
            std::swap(dev_idataPow, dev_odataPow);

            timer().endGpuTimer();

            hipMemcpy(odata, dev_odataPow, n * sizeof(int), hipMemcpyDeviceToHost);

            hipFree(dev_idataPow);
            hipFree(dev_bdataPow);
            hipFree(dev_edataPow);
            hipFree(dev_fdataPow);
            hipFree(dev_ddataPow);
            hipFree(dev_odataPow);

        }

    }
}